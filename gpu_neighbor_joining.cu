
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <assert.h>
#include <stdbool.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <climits>
#include <chrono>

using namespace std;
using namespace std::chrono;
#define TILE_WIDTH 20

const int MAX_TAXA = 2000;

void checkCudaError(hipError_t err)
{
    if (err != hipSuccess) {
        printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        exit(1);
    }
}


struct node {
    int node_name;
    struct node* leftChild;
    struct node* rightChild;
    struct node* parent;
    float distance_left, distance_right;
};
typedef struct node Node;

Node* Node_new_all(int s, Node* lChild, Node* rChild, float lDistance, float rDistance){
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = lChild;
    this_node->rightChild    = rChild;
    this_node->parent        = nullptr;
    this_node->distance_left = lDistance;  
    this_node->distance_right = rDistance;  
    lChild->parent      = this_node;
    rChild->parent      = this_node;
    return this_node;
}

Node* Node_new(int s) {
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = nullptr;
    this_node->rightChild    = nullptr;
    this_node->parent        = nullptr;
    this_node->distance_left = -1;  
    this_node->distance_right = -1;  
    return(this_node);
}

int readFromFile(float* dist_mat, char seq[MAX_TAXA], string filename, Node* nodes[MAX_TAXA]) {
    cout<<filename<<endl;
    ifstream infile(filename);

    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    infile.peek();
    int numRows = 0, numCols = 0;
    for (int i = 0; i < num_taxa; i++) {
        for (int j = 0; j < num_taxa; j++) {
            dist_mat[i*num_taxa + j] = 0;
        }
    }

    string s;
    int l_node = 0;
    while (!infile.eof() && numRows < num_taxa) {
        numCols = 0;
	    infile >> s;
        nodes[numRows] = Node_new(l_node);
	    infile.peek();
        while (infile.peek() != '\n' && numCols < numRows) {
            infile >> dist_mat[numRows*num_taxa + numCols];
            dist_mat[numCols*num_taxa + numRows] = dist_mat[numRows*num_taxa + numCols];
            numCols++;
        }
        infile.ignore(); // ignore newline character
        numRows++;
        l_node++;
    }

    infile.close();
    return num_taxa;
}

void printDistanceMatrix(float* dist_mat, int num_taxa, Node* nodes[MAX_TAXA]){
	cout<< "Num_taxa = " << num_taxa <<endl;
    for (int i = 0; i < num_taxa; i++) {
        if(nodes[i]==nullptr)        {
		    cout<<"Seq "<<i<<" = "<<"NULL" << " : ";
        }
        else {
		    cout<<"Seq "<<i<<" = "<<to_string(nodes[i]->node_name) << " : ";
        }
        for (int j = 0; j < num_taxa; j++) {
            cout << dist_mat[i*num_taxa + j] << " ";
        }
        cout << endl;
    }
}

void printTDMatrix(float TD_arr[MAX_TAXA], int num_taxa){
    for(int i=0 ; i<num_taxa; i++){
        cout<<TD_arr[i] << " " ;
    }
    cout<<endl;
}

///brief To dump values from Tree into a text file for graph generation
void traverseAndWrite(Node* node, ofstream& outfile) {
    if (node != NULL) {
        // Process the current node
        if(node->leftChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->leftChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_left <<"]"<<endl;

        }
        if(node->rightChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->rightChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_right <<"]"<<endl;

        }
        // Traverse the left subtree
        traverseAndWrite(node->leftChild, outfile);

        // Traverse the right subtree
        traverseAndWrite(node->rightChild, outfile);
    }
}



__global__ void gpu_nj_sum(int num_taxa, float* d_dist_mat, float* d_TD_arr, float* d_TB_min, Node** d_nodes, Node* d_temp_node, float* d_index1, float* d_index2, int nj_iter){

    /*
        Device variables needed - dist_mat, TD_arr
    */ 

    //int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int t_row = blockIdx.x*blockDim.x + threadIdx.y;
    int t_row_z = threadIdx.y;
    int t_col_og = threadIdx.x;
    int t_col = t_col_og;
    //int t_row = tid / num_taxa;
    //int t_col = tid % num_taxa;
    //__shared__ int index1, index2;
    //__shared__ int min_index, max_index;
    //__shared__ float delta_ij, limb_length_i, limb_length_j;
    //__shared__ float D_star_mat[32][2]; 
    //__shared__ float s_td_arr[32];

    //int index1, index2;
    //__shared__ int min_index;
    //__shared__ int max_index;
    //__shared__ float delta_ij;
    //__shared__ float limb_length_i;
    //__shared__ float limb_length_j;
    //__shared__ float min_D_star_mat[TILE_WIDTH][2];
    __shared__ int col_active[TILE_WIDTH];
    __shared__ int row_active[TILE_WIDTH];
    __shared__ float sum_tile[TILE_WIDTH][TILE_WIDTH];
    //float d_TD_arr[32];
    //__shared__ float min_row_mat[2][TILE_WIDTH][TILE_WIDTH];
    //int i, j;
    //float min_d_star_row;
    int col_tile_iter;
    int par_sum_iter = TILE_WIDTH / 2;
    //int n = num_taxa - nj_iter;

    // FIXME: OPT - can go down the column per thread
    // parallel sum possible 

        // initialize sum to 0
        
        sum_tile[t_row_z][t_col_og] = 0;

        if(t_col_og == 0){
            row_active[t_row_z] = -1;
            if(d_dist_mat[t_row] != -1 && t_row < num_taxa){
                row_active[t_row_z] = 1;
            }
        }
        __syncthreads();
        
        // loop needed to iterate over the columns of dist_mat
        for(col_tile_iter = 0; col_tile_iter < gridDim.x; col_tile_iter++) {
            t_col = t_col_og + col_tile_iter*TILE_WIDTH;
            // load which col is active amongst the current TILE_WIDTH
            if(t_row_z == 0) {
                col_active[t_col_og] = -1;
                if(d_dist_mat[t_col] != -1 && t_col < num_taxa){
                    col_active[t_col_og] = 1;
                }
            }
            __syncthreads();
            // find the sum corresponding to current TILE_WIDTH
            if(col_active[t_col_og] == 1  && row_active[t_row_z] == 1){
                sum_tile[t_row_z][t_col_og] += d_dist_mat[t_row*num_taxa + t_col];
                //printf("row %d of sum_tile added\n", t_row_z);
            }
        }
        __syncthreads();



    
        // store the calculated sum to d_TD_arr
        
        if(t_col_og == 0){
            for(par_sum_iter = 1; par_sum_iter < TILE_WIDTH; par_sum_iter++){
                sum_tile[t_row_z][0] += sum_tile[t_row_z][t_col_og + par_sum_iter];
            }
            if(row_active[t_row_z] == 1){
                d_TD_arr[t_row] = sum_tile[t_row_z][0];
            }
            else {
                d_TD_arr[t_row] = -1;
            }
        }
        
        __syncthreads();

// d_TD_arr should have all the values of the rows

};




__global__ void gpu_nj_min(int num_taxa, float* d_dist_mat, float* d_TD_arr, float* d_TB_min, Node** d_nodes, Node* d_temp_node, float* d_index1, float* d_index2, int nj_iter){
       

    //int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int t_row = blockIdx.x*blockDim.x + threadIdx.y;
    int t_row_z = threadIdx.y;
    int t_col_og = threadIdx.x;
    int t_col = t_col_og;

    //int index1, index2;
    //__shared__ int min_index;
    //__shared__ int max_index;
    //__shared__ float delta_ij;
    //__shared__ float limb_length_i;
    //__shared__ float limb_length_j;
    //__shared__ float min_D_star_mat[TILE_WIDTH][2];
    __shared__ int col_active[TILE_WIDTH];
    __shared__ int row_active[TILE_WIDTH];
    //__shared__ float sum_tile[TILE_WIDTH][TILE_WIDTH];
    //float d_TD_arr[32];
    __shared__ float min_row_mat[2][TILE_WIDTH][TILE_WIDTH];
    //int i, j;
    //float min_d_star_row;
    int col_tile_iter;
    //int par_sum_iter = TILE_WIDTH / 2;
    int n = num_taxa - nj_iter;



        if(t_col_og == 0){
            row_active[t_row_z] = -1;
            if(d_dist_mat[t_row] != -1 && t_row < num_taxa){
                row_active[t_row_z] = 1;
            }
        }
        __syncthreads();

 
        // initialize min_row_mat with INT_MAX
        min_row_mat[0][t_row_z][t_col_og] = INT_MAX;
        min_row_mat[1][t_row_z][t_col_og] = t_col_og;

        float curr_value, calc_value, curr_index;
        curr_value = INT_MAX;
        calc_value = INT_MAX;
        curr_index = t_col_og;
        __syncthreads();

        // for each tile iterate and find the min of the D_star similar to sum
        
        for(col_tile_iter = 0; col_tile_iter < gridDim.x; col_tile_iter++) {
            t_col = t_col_og + col_tile_iter*TILE_WIDTH;
            // load which col is active amongst the current TILE_WIDTH
            if(t_row_z == 0) {
                col_active[t_col_og] = -1;
                if(d_dist_mat[t_col] != -1 && t_col < num_taxa){
                    col_active[t_col_og] = 1;
                }
            }
            __syncthreads();
            //  find the min of current and iter 
            if(col_active[t_col_og] == 1 && row_active[t_row_z] == 1 && t_row != t_col){
                // calculate the D_star value for the elem
                // compare with the existing min
                // replace if less
                calc_value = (n - 2) * d_dist_mat[t_row*num_taxa + t_col] - d_TD_arr[t_row] - d_TD_arr[t_col];
                if(calc_value < curr_value){
                    curr_value = calc_value;
                    curr_index = t_col;
                }
            }
        }
        // store the curr_index and curr_value for that thread
        min_row_mat[0][t_row_z][t_col_og] = curr_value;
        min_row_mat[1][t_row_z][t_col_og] = curr_index;
        __syncthreads();

        
        int min_iter;
        // find the min of the row and its index
        if(t_col_og == 0 && row_active[t_row_z] == 1){
            for(min_iter = 1; min_iter < TILE_WIDTH; min_iter++){
                if(min_row_mat[0][t_row_z][min_iter] < curr_value) {
                   curr_value = min_row_mat[0][t_row_z][min_iter];
                   curr_index = min_row_mat[1][t_row_z][min_iter];
                }
            }
            min_row_mat[0][t_row_z][0] = curr_value;
            min_row_mat[1][t_row_z][0] = curr_index;
        }
        __syncthreads();
        // PRINT THE MIN_ROW_MAT[0] values


        float min_row_index, min_col_index;
        // find the min of all rows and the index pair
        if(t_row_z == 0 && t_col_og == 0){
            curr_value = min_row_mat[0][0][0];
            min_row_index = t_row;
            min_col_index = min_row_mat[1][0][0];
            for(min_iter = 1; min_iter < TILE_WIDTH; min_iter++){
                if(min_row_mat[0][min_iter][0] < curr_value){
                    curr_value = min_row_mat[0][min_iter][0];
                    min_col_index = min_row_mat[1][min_iter][0];
                    min_row_index = t_row + min_iter;
                }
            }
            d_TB_min[blockIdx.x] = curr_value;
            d_TB_min[gridDim.x + blockIdx.x] = min_row_index; // row
            d_TB_min[2*gridDim.x + blockIdx.x] = min_col_index; // col
        }
        __syncthreads();

/*
if(t_row == 0 && t_col_og == 0) {
    printf("d_TB_min = %lf, min_row_index = %lf, min_col_index = %lf\n", curr_value, min_row_index, min_col_index);
}
*/

};


__global__ void gpu_nj_update(int num_taxa, float* d_dist_mat, float* d_TD_arr, float* d_TB_min, Node** d_nodes, Node* d_temp_node, float* d_index1, float* d_index2, int nj_iter){

    //int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int t_row = blockIdx.x*blockDim.x + threadIdx.y;
    int t_row_z = threadIdx.y;
    int t_col_og = threadIdx.x;
    int t_col = t_col_og;

    int index1, index2;
    __shared__ int min_index;
    __shared__ int max_index;
    __shared__ float delta_ij;
    __shared__ float limb_length_i;
    __shared__ float limb_length_j;
    //__shared__ float min_D_star_mat[TILE_WIDTH][2];
    //__shared__ int col_active[TILE_WIDTH];
    //__shared__ int row_active[TILE_WIDTH];
    //__shared__ float sum_tile[TILE_WIDTH][TILE_WIDTH];
    //float d_TD_arr[32];
    //__shared__ float min_row_mat[2][TILE_WIDTH][TILE_WIDTH];
    //int i, j;
    //float min_d_star_row;
    //int col_tile_iter;
    //int par_sum_iter = TILE_WIDTH / 2;
    int n = num_taxa - nj_iter;
                       

    int min_iter;
    float curr_value; 

        // reduce the d_TB_min
        if(t_row_z == 0 && t_col_og == 0){
            curr_value = d_TB_min[0];
            index1 = d_TB_min[gridDim.x];
            index2 = d_TB_min[2*gridDim.x];
            for(min_iter = 1; min_iter < gridDim.x; min_iter++){
                if(d_TB_min[min_iter] < curr_value){
                    curr_value = d_TB_min[min_iter];
                    index1 = d_TB_min[gridDim.x + min_iter];
                    index2 = d_TB_min[2*gridDim.x + min_iter];
                }
            }
        }        
        __syncthreads();
                                     
                                     


        if(t_row_z == 0 && t_col_og == 0) {
            min_index = (index1 < index2) ? index1 : index2;
            max_index = (index1 < index2) ? index2 : index1;
            delta_ij = (d_TD_arr[min_index] - d_TD_arr[max_index]) / (n-2);
            limb_length_i = (d_dist_mat[min_index*num_taxa + max_index] + delta_ij)/2.0;
            limb_length_j = (d_dist_mat[min_index*num_taxa + max_index] - delta_ij)/2.0;
        }
        __syncthreads();


        //updateDistanceMatrix(d_dist_mat,num_taxa, min_index, max_index);

        // update the distance matrix parallely with new values at max index
        t_col = t_row;
    
        if(t_col_og == 0 && t_row != min_index && t_row != max_index && t_row < num_taxa){
            d_dist_mat[t_row*num_taxa + max_index] = (d_dist_mat[t_row*num_taxa + min_index] + d_dist_mat[t_row*num_taxa + max_index] - d_dist_mat[min_index*num_taxa + max_index]) / 2;
            d_dist_mat[max_index*num_taxa + t_row] = d_dist_mat[t_row*num_taxa + max_index];
        }
        __syncthreads();


        // turn min_index to -1,
        if(t_row == 0 && t_col_og == 0){
            d_dist_mat[min_index*num_taxa] = -1;
            d_dist_mat[min_index] = -1;
         
            d_temp_node->node_name      = nj_iter;
            d_temp_node->leftChild      = d_nodes[min_index];
            d_temp_node->rightChild     = d_nodes[max_index];
            d_temp_node->parent         = nullptr;
            d_temp_node->distance_left  = limb_length_i;
            d_temp_node->distance_right = limb_length_j;

            // should not create a new node in GPU, rather just change the values of existing array
            d_nodes[max_index] = d_temp_node;
            d_nodes[min_index] = nullptr;
        }
        __syncthreads();
        
             
};



int main() {
    
    //string filename = "./examples/evolution.in";
    //string filename = "./examples/INGI2368.in";
    string filename = "./scripting/IN1000.in";
    ifstream infile(filename);
    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    //float dist_mat[num_taxa*num_taxa];
    float* dist_mat;
    dist_mat = (float *)malloc(num_taxa*num_taxa*sizeof(float));
    float* d_dist_mat;
    char seq[num_taxa];
    Node* nodes[num_taxa];
    readFromFile(dist_mat, seq, filename, nodes);
    Node** d_nodes;
    Node* d_temp_node;
    printDistanceMatrix(dist_mat, num_taxa, nodes);
    //int index1, index2;
    //int min_index, max_index;
    //float delta_ij, limb_length_i, limb_length_j;
    //int n;
    //float TD_arr[num_taxa];
    float* d_TD_arr;
    float* d_TB_min;
    float* d_index1;
    float* d_index2;
    int num_TB = (num_taxa + TILE_WIDTH - 1) / TILE_WIDTH;

    // allocate memory and copy the variables to GPU, 
    // launch kernel
    // copy the variables to CPU
    // free GPU memory

    printf("*** Allocating GPU memory ***\n");
    hipMalloc((void**)(&d_dist_mat), num_taxa*num_taxa*(sizeof(float)));
    hipMalloc((void**)(&d_TD_arr), num_taxa*(sizeof(float)));
    hipMalloc((void**)(&d_TB_min), 3*num_TB*(sizeof(float)));
    hipMalloc((void**)(&d_index1), sizeof(float));
    hipMalloc((void**)(&d_index2), sizeof(float));
    hipMalloc((void**)(&d_nodes), num_taxa*(sizeof(Node)));
    hipMalloc((void**)(&d_temp_node), sizeof(Node));
    printf("*** Allocating GPU memory complete ***\n\n");

    printf("*** Copying to GPU memory ***\n");
    checkCudaError(hipMemcpy(d_dist_mat, dist_mat, num_taxa*num_taxa*(sizeof(float)), hipMemcpyHostToDevice));    
    //cudaMemcpy(&d_TD_arr, &TD_arr, num_taxa*(sizeof(float)), cudaMemcpyHostToDevice);
    //cudaMemcpy(&d_TB_min, &d_TB_min, 3*num_TB*(sizeof(float)), cudaMemcpyHostToDevice);
    //cudaMemcpy(&d_nodes, &nodes, num_taxa*(sizeof(Node)), cudaMemcpyHostToDevice);
    printf("*** Copying to GPU memory complete ***\n\n");

    // Parallelize GPU set grid, block and call kernel
    dim3 blocksize(TILE_WIDTH,TILE_WIDTH);
    dim3 gridsize((num_taxa + TILE_WIDTH - 1) / TILE_WIDTH);
   
    //printf("Launching kernel with griddim: %d, %d, %d\n", gridDim.x, gridDim.y, gridDim.z);
    //printf("Launching kernel with blockDim: %d, %d, %d\n", blockDim.x, blockDim.y, blockDim.z);
    
    int nj_iter = 0;

    auto start_time = high_resolution_clock::now();     
    for(nj_iter = 0; nj_iter < num_taxa - 2; nj_iter++) {
        gpu_nj_sum<<<gridsize, blocksize>>>(num_taxa, d_dist_mat, d_TD_arr, d_TB_min,  d_nodes, d_temp_node, d_index1, d_index2, nj_iter);
        hipDeviceSynchronize();    
        gpu_nj_min<<<gridsize, blocksize>>>(num_taxa, d_dist_mat, d_TD_arr, d_TB_min,  d_nodes, d_temp_node, d_index1, d_index2, nj_iter);
        hipDeviceSynchronize();    
        gpu_nj_update<<<gridsize, blocksize>>>(num_taxa, d_dist_mat, d_TD_arr, d_TB_min,  d_nodes, d_temp_node, d_index1, d_index2, nj_iter);
        hipDeviceSynchronize();    
    }
    
 
    auto end_time = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end_time - start_time);
    printf("### \n Elapsed Time %" PRId64 "\n###\n", duration.count());
   
 
    //checkCudaError(cudaGetLastError());
    printf("***  GPU computation complete ***\n");

    checkCudaError(hipMemcpy(dist_mat, d_dist_mat, num_taxa*num_taxa*sizeof(float), hipMemcpyDeviceToHost));
    printf("*** Transferring data from Device to Host complete ***\n");
    //checkCudaError(cudaMemcpy(TD_arr, d_TD_arr, num_taxa*sizeof(float), cudaMemcpyDeviceToHost));
    printf("*** Transferring data from Device to Host complete ***\n");
    //checkCudaError(cudaMemcpy(nodes, *d_nodes, num_taxa*sizeof(Node), cudaMemcpyDeviceToHost));
    printf("*** Transferring data from Device to Host complete ***\n");

    int final_index1 = -1;
    int final_index2 = -1;

    int i;
    for(i=0 ; i<num_taxa ; i++) {
        if(dist_mat[i*num_taxa + 0]!=-1)
        {
            if(final_index1==-1)
                final_index1 = i;
            else
                final_index2 = i;
        }
    } 

    int root_node_name = i;
    cout<<to_string(root_node_name)<<endl;
    Node* root = Node_new_all(root_node_name, nodes[final_index1], nodes[final_index2], dist_mat[final_index1*num_taxa + final_index2]/2.0, dist_mat[final_index1*num_taxa + final_index2]/2.0 );
    
    printf("*** Final node computed ***\n");
    printDistanceMatrix(dist_mat, num_taxa, nodes);

    ofstream outfile("g.gv"); // open the output file
    if (!outfile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    outfile << "digraph {" << endl;
    traverseAndWrite(root, outfile);
    outfile << "}" << endl;
    outfile.close();

    //FIXME:free the gpu memory and all variables

    hipFree(d_dist_mat);
    hipFree(d_TD_arr);
    hipFree(d_TB_min);
    hipFree(d_nodes);
    hipFree(d_temp_node);

    return 0;
}
