
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <assert.h>
#include <stdbool.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <climits>

using namespace std;

const int MAX_TAXA = 100;

struct node {
    int node_name;
    struct node* leftChild;
    struct node* rightChild;
    struct node* parent;
    double distance_left, distance_right;
};
typedef struct node Node;

Node* Node_new_all(int s, Node* lChild, Node* rChild, double lDistance, double rDistance){
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = lChild;
    this_node->rightChild    = rChild;
    this_node->parent        = nullptr;
    this_node->distance_left = lDistance;  
    this_node->distance_right = rDistance;  
    lChild->parent      = this_node;
    rChild->parent      = this_node;
    return this_node;
}

Node* Node_new(int s) {
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = nullptr;
    this_node->rightChild    = nullptr;
    this_node->parent        = nullptr;
    this_node->distance_left = -1;  
    this_node->distance_right = -1;  
    return(this_node);
}

int readFromFile(double dist_mat[MAX_TAXA][MAX_TAXA], char seq[MAX_TAXA], string filename, Node* nodes[MAX_TAXA]) {
    cout<<filename<<endl;
    ifstream infile(filename);

    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    infile.peek();
    int numRows = 0, numCols = 0;
    //Initialize Distance Matrix to 0
    for (int i = 0; i < num_taxa; i++) {
        for (int j = 0; j < num_taxa; j++) {
            dist_mat[i][j] = 0;
        }
    }

    while (!infile.eof() && numRows < num_taxa) {
        numCols = 0;
	    infile >> seq[numRows];
        nodes[numRows] = Node_new({seq[numRows]});
	    infile.peek();
        while (infile.peek() != '\n' && numCols < numRows) {
            infile >> dist_mat[numRows][numCols];
            dist_mat[numCols][numRows] = dist_mat[numRows][numCols];
            numCols++;
        }
        infile.ignore(); // ignore newline character
        numRows++;
    }

    infile.close();
    return num_taxa;
}

void printDistanceMatrix(double dist_mat[MAX_TAXA][MAX_TAXA], int num_taxa, Node* nodes[MAX_TAXA]){
	cout<< "Num_taxa = " << num_taxa <<endl;
    for (int i = 0; i < num_taxa; i++) {
        if(nodes[i]==nullptr)        {
		    cout<<"Seq "<<i<<" = "<<"NULL" << " : ";
        }
        else {
		    cout<<"Seq "<<i<<" = "<<to_string(nodes[i]->node_name) << " : ";
        }
        for (int j = 0; j < num_taxa; j++) {
            cout << dist_mat[i][j] << " ";
        }
        cout << endl;
    }
}

void printTDMatrix(double TD_arr[MAX_TAXA], int num_taxa){
    for(int i=0 ; i<num_taxa; i++){
        cout<<TD_arr[i] << " " ;
    }
    cout<<endl;
}

///brief To dump values from Tree into a text file for graph generation
void traverseAndWrite(Node* node, ofstream& outfile) {
    if (node != NULL) {
        // Process the current node
        if(node->leftChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->leftChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_left <<"]"<<endl;

        }
        if(node->rightChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->rightChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_right <<"]"<<endl;

        }
        // Traverse the left subtree
        traverseAndWrite(node->leftChild, outfile);

        // Traverse the right subtree
        traverseAndWrite(node->rightChild, outfile);
    }
}



__global__ void gpu_nj(int num_taxa, double* d_dist_mat, double* d_TD_arr, Node** d_nodes, Node* d_temp_node){

    /*
        Device variables needed - dist_mat, TD_arr
    */ 

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //int t_row = tid / num_taxa;
    //int t_col = tid % num_taxa;
    __shared__ int index1, index2;
    __shared__ int min_index, max_index;
    __shared__ double delta_ij, limb_length_i, limb_length_j;
    int n;
    int i;
    double sum, min_d_star_row;

    __shared__ double D_star_mat[100][2]; // declare in shared memory later
    __shared__ double s_td_arr[100];

    // load TD_arr in shared memory
    if(tid < num_taxa);
        s_td_arr[tid] = d_TD_arr[tid];
    
   
    // OPT - can go down the column per thread
    // parallel sum possible 
    for(i=0 ; i<num_taxa-2; i++) {
        n = num_taxa - i;
        //totalDistance(dist_mat, num_taxa, TD_arr);
        // GPU implementation of totalDistance
        if(tid < num_taxa) {
            if(d_dist_mat[tid*num_taxa] != -1) {
                sum=0;
                for (int k = 0; k < num_taxa; k++) {
                    if(d_dist_mat[k*num_taxa] != -1){
                        sum += d_dist_mat[tid*num_taxa + k];
                    }
                }
                s_td_arr[tid] = sum;
            } else{
                s_td_arr[tid] = -1;
            }
        }
        __syncthreads();


        //find_closest_pair(d_dist_mat,num_taxa, TD_arr, index1, index2);
        // GPU code for find_closest_pair
        min_d_star_row = INT_MAX;
        
        if(tid < num_taxa) {
            D_star_mat[tid][0] = INT_MAX;
            if(d_dist_mat[tid*num_taxa] != -1) {
                for (int j = tid + 1; j < num_taxa; j++) {
                    if(d_dist_mat[j*num_taxa] != -1){
                        min_d_star_row = (num_taxa - 2) * d_dist_mat[tid*num_taxa + j] - s_td_arr[tid] - s_td_arr[j];
                        if (min_d_star_row < D_star_mat[tid][0]) {
                            D_star_mat[tid][0] = min_d_star_row;
                            D_star_mat[tid][1] = j;
                            
                            }
                        }
                    }
                } else {
                    D_star_mat[tid][0] = INT_MAX;
                }
            }
        __syncthreads();

        // find the index pair which has absolute min among the d_star
        if(tid == 0) {
            min_d_star_row = INT_MAX;
            for (int j = 0; j < num_taxa; j++) {
                if(D_star_mat[j][0] < min_d_star_row){
                    min_d_star_row = D_star_mat[j][0];
                    index1 = j;
                    index2 = D_star_mat[tid][1];
                }
            }
        }


        if(tid == 0) {
            min_index = (index1 < index2) ? index1 : index2;
            max_index = (index1 < index2) ? index2 : index1;
            delta_ij = (s_td_arr[min_index] - s_td_arr[max_index]) / (n-2);
            limb_length_i = (d_dist_mat[min_index*num_taxa + max_index] + delta_ij)/2.0;
            limb_length_j = (d_dist_mat[min_index*num_taxa + max_index] - delta_ij)/2.0;
        }


        //updateDistanceMatrix(d_dist_mat,num_taxa, min_index, max_index);

        // update the distance matrix parallely with new values at max index

        if((tid < num_taxa) && (tid != min_index) && (tid != max_index)) {
            d_dist_mat[max_index*num_taxa + tid] = (d_dist_mat[min_index*num_taxa + tid] + d_dist_mat[max_index*num_taxa + tid] - d_dist_mat[min_index*num_taxa + max_index]) / 2;
            d_dist_mat[tid*num_taxa + max_index] = d_dist_mat[max_index*num_taxa + tid];
        }


        // turn min_index to -1,
        if(tid == 0){
            d_dist_mat[min_index*num_taxa] = -1;
            d_dist_mat[min_index] = -1;
         
            d_temp_node->node_name      = i;
            d_temp_node->leftChild      = nullptr;
            d_temp_node->rightChild     = nullptr;
            d_temp_node->parent         = nullptr;
            d_temp_node->distance_left  = -1;
            d_temp_node->distance_right = -1;

            // should not create a new node in GPU, rather just change the values of existing array
            d_nodes[max_index] = d_temp_node;
            d_nodes[min_index] = nullptr;
        }
        __syncthreads();
    }

    // Copying the TD_arr back to GPU global memory
    if(tid < num_taxa);
        d_TD_arr[tid] = s_td_arr[tid];

};



int main() {
    
    string filename = "./examples/INGI2368.in";
    ifstream infile(filename);
    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    double dist_mat[MAX_TAXA][MAX_TAXA];
    double d_dist_mat[num_taxa*num_taxa];
    char seq[num_taxa];
    Node* nodes[num_taxa];
    readFromFile(dist_mat, seq, filename, nodes);
    Node* d_nodes[num_taxa];
    Node* d_temp_node;
    printDistanceMatrix(dist_mat, num_taxa, nodes);
    //int index1, index2;
    //int min_index, max_index;
    //double delta_ij, limb_length_i, limb_length_j;
    //int n;
    double TD_arr[num_taxa];
    double d_TD_arr[num_taxa];


    // allocate memory and copy the variables to GPU, 
    // launch kernel
    // copy the variables to CPU
    // free GPU memory

    printf("*** Allocating GPU memory ***\n");
    hipMalloc((void**)(&d_dist_mat), num_taxa*num_taxa*(sizeof(double)));
    hipMalloc((void**)(&d_TD_arr), num_taxa*(sizeof(double)));
    hipMalloc((void**)(&d_nodes), num_taxa*(sizeof(Node)));
    hipMalloc((void**)(&d_temp_node), sizeof(Node));
    printf("*** Allocating GPU memory complete ***\n\n");

    printf("*** Copying to GPU memory ***\n");
    hipMemcpy(d_dist_mat, dist_mat, num_taxa*num_taxa*(sizeof(double)), hipMemcpyHostToDevice);    
    //cudaMemcpy(&d_TD_arr, &TD_arr, num_taxa*(sizeof(double)), cudaMemcpyHostToDevice);
    //cudaMemcpy(&d_nodes, &nodes, num_taxa*(sizeof(double)), cudaMemcpyHostToDevice);
    printf("*** Copying to GPU memory complete ***\n\n");

    // Parallelize GPU set grid, block and call kernel
    dim3 blockDim(32);
    dim3 gridDim(ceil(num_taxa / 32));
    
    gpu_nj<<<gridDim, blockDim>>>(num_taxa, d_dist_mat, d_TD_arr, d_nodes, d_temp_node);
    
    printf("***  GPU computation complete ***\n");
    hipMemcpy(dist_mat, d_dist_mat, num_taxa*num_taxa*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(TD_arr, d_TD_arr, num_taxa*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(nodes, d_nodes, num_taxa*sizeof(double), hipMemcpyDeviceToHost);
    printf("*** Transferring data from Device to Host complete ***\n");

    int final_index1 = -1;
    int final_index2 = -1;

    int i;
    for(i=0 ; i<num_taxa ; i++) {
        if(dist_mat[i][0]!=-1)
        {
            if(final_index1==-1)
                final_index1 = i;
            else
                final_index2 = i;
        }
    } 

    int root_node_name = i;
    cout<<to_string(root_node_name)<<endl;
    Node* root = Node_new_all(root_node_name, nodes[final_index1], nodes[final_index2], dist_mat[final_index1][final_index2]/2.0, dist_mat[final_index1][final_index2]/2.0 );
    
    printf("*** Final node computed ***\n");
    printDistanceMatrix(dist_mat, num_taxa, nodes);

    ofstream outfile("g.gv"); // open the output file
    if (!outfile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    outfile << "digraph {" << endl;
    traverseAndWrite(root, outfile);
    outfile << "}" << endl;
    outfile.close();


    return 0;
}
