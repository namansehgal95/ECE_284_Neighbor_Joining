
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <assert.h>
#include <stdbool.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <climits>

using namespace std;

const int MAX_TAXA = 100;

struct node {
    public:
    int node_name;
    struct node* leftChild;
    struct node* rightChild;
    struct node* parent;
    double distance_left, distance_right;
};
typedef struct node Node;

Node* Node_new_all(int s, Node* lChild, Node* rChild, double lDistance, double rDistance){
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = lChild;
    this_node->rightChild    = rChild;
    this_node->parent        = nullptr;
    this_node->distance_left = lDistance;  
    this_node->distance_right = rDistance;  
    lChild->parent      = this_node;
    rChild->parent      = this_node;
    return this_node;
}

Node* Node_new(int s) {
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = nullptr;
    this_node->rightChild    = nullptr;
    this_node->parent        = nullptr;
    this_node->distance_left = -1;  
    this_node->distance_right = -1;  
    return(this_node);
}

int readFromFile(double arr[MAX_TAXA][MAX_TAXA], char seq[MAX_TAXA], string filename, Node* nodes[MAX_TAXA]) {
    cout<<filename<<endl;
    ifstream infile(filename);

    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    infile.peek();
    int numRows = 0, numCols = 0;
    //Initialize Distance Matrix to 0
    for (int i = 0; i < num_taxa; i++) {
        for (int j = 0; j < num_taxa; j++) {
            arr[i][j] = 0;
        }
    }

    while (!infile.eof() && numRows < num_taxa) {
        numCols = 0;
	    infile >> seq[numRows];
        nodes[numRows] = Node_new({seq[numRows]});
	    infile.peek();
        while (infile.peek() != '\n' && numCols < numRows) {
            infile >> arr[numRows][numCols];
            arr[numCols][numRows] = arr[numRows][numCols];
            numCols++;
        }
        infile.ignore(); // ignore newline character
        numRows++;
    }

    infile.close();
    return num_taxa;
}

void printDistanceMatrix(double arr[MAX_TAXA][MAX_TAXA], int num_taxa, Node* nodes[MAX_TAXA]){
	cout<< "Num_taxa = " << num_taxa <<endl;
    for (int i = 0; i < num_taxa; i++) {
        if(nodes[i]==nullptr)        {
		    cout<<"Seq "<<i<<" = "<<"NULL" << " : ";
        }
        else {
		    cout<<"Seq "<<i<<" = "<<to_string(nodes[i]->node_name) << " : ";
        }
        for (int j = 0; j < num_taxa; j++) {
            cout << arr[i][j] << " ";
        }
        cout << endl;
    }
}

void printTDMatrix(double TD_arr[MAX_TAXA], int num_taxa){
    for(int i=0 ; i<num_taxa; i++){
        cout<<TD_arr[i] << " " ;
    }
    cout<<endl;
}

///brief To dump values from Tree into a text file for graph generation
void traverseAndWrite(Node* node, ofstream& outfile) {
    if (node != NULL) {
        // Process the current node
        if(node->leftChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->leftChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_left <<"]"<<endl;

        }
        if(node->rightChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->rightChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_right <<"]"<<endl;

        }
        // Traverse the left subtree
        traverseAndWrite(node->leftChild, outfile);

        // Traverse the right subtree
        traverseAndWrite(node->rightChild, outfile);
    }
}


void totalDistance(double arr[MAX_TAXA][MAX_TAXA], int num_taxa, double TD_arr[MAX_TAXA]){

    // total NUM_TAXA threads,
    // each thread will be calculating the sum for TD_arr[i]
    for(int i=0; i<num_taxa; i++){
        double sum=0;
        if(arr[i][0]!=-1) {
            for (int k = 0; k < num_taxa; k++) {
                if(arr[k][0]!=-1){
                    sum += arr[i][k];
                }
            }
            TD_arr[i] = sum;
        } else{
            TD_arr[i] = -1;
        }
    }
}

/// @brief Calculate indexes with minimum D_star and store in array variable pair
// Check if -1
void find_closest_pair(double arr[MAX_TAXA][MAX_TAXA], int num_taxa, double TD_arr[MAX_TAXA], int& index1, int& index2) {
    // less than num_taxa*num_taxa/2 threads used
    // change this code:
    // first create a D_star matrix in shared memory - each thread independently
    // Then find the minimum of the D_star again using parallelism using parallel_red
    
    double min_distance = INT_MAX;
    for (int i = 0; i < num_taxa; i++) {
        if(arr[i][0]!=-1) {
            for (int j = i + 1; j < num_taxa; j++) {
                if(arr[j][0]!=-1){
                    double D_star = (num_taxa - 2) * arr[i][j] - TD_arr[i] - TD_arr[j];
                    if (D_star < min_distance) {
                        min_distance = D_star;
                        index1 = i;
                        index2 = j;
                    }
                }
            }
        }
    }
    //return min_distance / (num_taxa - 2);
}

//Pending
void updateDistanceMatrix(double arr[MAX_TAXA][MAX_TAXA], int num_taxa, int min_index, int max_index) {
    // update the distance matrix parallely with new values
    for (int k = 0; k < num_taxa; k++) {
        if (k != min_index && k != max_index) {
            arr[max_index][k] = ( arr[min_index][k] + arr[max_index][k] - arr[min_index][max_index]) / 2;
            arr[k][max_index] = arr[max_index][k];
        }
    }
    arr[min_index][0] = arr[0][min_index] = -1;
}

int main() {
    
    string file_name = "./examples/INGI2368.in";
    double arr[MAX_TAXA][MAX_TAXA];
    char seq[MAX_TAXA];
    Node* nodes[MAX_TAXA];
    //int num_taxa = read_DM_file(arr, seq, file_name, nodes);
    int num_taxa = readFromFile(arr, seq, file_name, nodes);
    printDistanceMatrix(arr, num_taxa, nodes);
    int index1, index2;
    int min_index, max_index;
    double delta_ij, limb_length_i, limb_length_j;
    int n;

    double TD_arr[MAX_TAXA];
    for(int i=0 ; i<num_taxa -2; i++) {
        n = num_taxa - i;
        // Parallelize GPU
        totalDistance(arr, num_taxa, TD_arr);
        printTDMatrix(TD_arr, num_taxa);
        find_closest_pair(arr,num_taxa, TD_arr, index1, index2);
        //}
        
        min_index = min(index1, index2);
        max_index = max(index1, index2);
        delta_ij = (TD_arr[min_index] - TD_arr[max_index]) / (n-2);
        limb_length_i = (arr[min_index][max_index] + delta_ij)/2.0;
        limb_length_j = (arr[min_index][max_index] - delta_ij)/2.0;
        updateDistanceMatrix(arr,num_taxa, min_index, max_index);
        int new_node_name = i;
        cout<<to_string(new_node_name)<<endl;
        Node* temp = Node_new_all(new_node_name, nodes[min_index], nodes[max_index], limb_length_i, limb_length_j );
        nodes[max_index] = temp;
        nodes[min_index] = nullptr;
        printDistanceMatrix(arr, num_taxa, nodes);
    }
    int final_index1 = -1;
    int final_index2 = -1;

    int i;
    for(i=0 ; i<num_taxa ; i++) {
        if(arr[i][0]!=-1)
        {
            if(final_index1==-1)
                final_index1 = i;
            else
                final_index2 = i;
        }
    } 

    int root_node_name = i;
    cout<<to_string(root_node_name)<<endl;
    Node* root = Node_new_all(root_node_name, nodes[final_index1], nodes[final_index2], arr[final_index1][final_index2]/2.0, arr[final_index1][final_index2]/2.0 );

    // cout<<nodes[0]->node_name<<" "<<nodes[1]->node_name;
    
    ofstream outfile("g.gv"); // open the output file
    if (!outfile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    outfile << "digraph {" << endl;
    traverseAndWrite(root, outfile);
    outfile << "}" << endl;
    outfile.close();


    return 0;
}
