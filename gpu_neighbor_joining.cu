#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <assert.h>
#include <stdbool.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <climits>

using namespace std;

const int MAX_TAXA = 100;

struct node {
    public:
    int node_name;
    struct node* leftChild;
    struct node* rightChild;
    struct node* parent;
    double distance_left, distance_right;
};
typedef struct node Node;

Node* Node_new_all(int s, Node* lChild, Node* rChild, double lDistance, double rDistance){
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = lChild;
    this_node->rightChild    = rChild;
    this_node->parent        = nullptr;
    this_node->distance_left = lDistance;  
    this_node->distance_right = rDistance;  
    lChild->parent      = this_node;
    rChild->parent      = this_node;
    return this_node;
}

Node* Node_new(int s) {
    Node* this_node = (Node*)malloc(sizeof(Node));
    this_node->node_name     = s;
    this_node->leftChild     = nullptr;
    this_node->rightChild    = nullptr;
    this_node->parent        = nullptr;
    this_node->distance_left = -1;  
    this_node->distance_right = -1;  
    return(this_node);
}

int readFromFile(double dist_mat[MAX_TAXA][MAX_TAXA], char seq[MAX_TAXA], string filename, Node* nodes[MAX_TAXA]) {
    cout<<filename<<endl;
    ifstream infile(filename);

    if (!infile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    int num_taxa;
    infile >> num_taxa;
    infile.peek();
    int numRows = 0, numCols = 0;
    //Initialize Distance Matrix to 0
    for (int i = 0; i < num_taxa; i++) {
        for (int j = 0; j < num_taxa; j++) {
            dist_mat[i][j] = 0;
        }
    }

    while (!infile.eof() && numRows < num_taxa) {
        numCols = 0;
	    infile >> seq[numRows];
        nodes[numRows] = Node_new({seq[numRows]});
	    infile.peek();
        while (infile.peek() != '\n' && numCols < numRows) {
            infile >> dist_mat[numRows][numCols];
            dist_mat[numCols][numRows] = dist_mat[numRows][numCols];
            numCols++;
        }
        infile.ignore(); // ignore newline character
        numRows++;
    }

    infile.close();
    return num_taxa;
}

void printDistanceMatrix(double dist_mat[MAX_TAXA][MAX_TAXA], int num_taxa, Node* nodes[MAX_TAXA]){
	cout<< "Num_taxa = " << num_taxa <<endl;
    for (int i = 0; i < num_taxa; i++) {
        if(nodes[i]==nullptr)        {
		    cout<<"Seq "<<i<<" = "<<"NULL" << " : ";
        }
        else {
		    cout<<"Seq "<<i<<" = "<<to_string(nodes[i]->node_name) << " : ";
        }
        for (int j = 0; j < num_taxa; j++) {
            cout << dist_mat[i][j] << " ";
        }
        cout << endl;
    }
}

void printTDMatrix(double TD_arr[MAX_TAXA], int num_taxa){
    for(int i=0 ; i<num_taxa; i++){
        cout<<TD_arr[i] << " " ;
    }
    cout<<endl;
}

///brief To dump values from Tree into a text file for graph generation
void traverseAndWrite(Node* node, ofstream& outfile) {
    if (node != NULL) {
        // Process the current node
        if(node->leftChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->leftChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_left <<"]"<<endl;

        }
        if(node->rightChild!=nullptr) {
            outfile << "\"" << to_string(node->node_name) << "\" ";
            outfile << "->" << "\"" << to_string(node->rightChild->node_name) << "\" ";
            outfile <<"[taillabel = " <<fixed << setprecision(2) << node->distance_right <<"]"<<endl;

        }
        // Traverse the left subtree
        traverseAndWrite(node->leftChild, outfile);

        // Traverse the right subtree
        traverseAndWrite(node->rightChild, outfile);
    }
}


void totalDistance(double dist_mat[MAX_TAXA][MAX_TAXA], int num_taxa, double TD_arr[MAX_TAXA]){

    // total NUM_TAXA threads,
    // each thread will be calculating the sum for TD_arr[i]
    for(int i=0; i<num_taxa; i++){
        double sum=0;
        if(dist_mat[i][0]!=-1) {
            for (int k = 0; k < num_taxa; k++) {
                if(dist_mat[k][0]!=-1){
                    sum += dist_mat[i][k];
                }
            }
            TD_arr[i] = sum;
        } else{
            TD_arr[i] = -1;
        }
    }
}

/// @brief Calculate indexes with minimum D_star and store in array variable pair
// Check if -1
void find_closest_pair(double dist_mat[MAX_TAXA][MAX_TAXA], int num_taxa, double TD_arr[MAX_TAXA], int& index1, int& index2) {
    // less than num_taxa*num_taxa/2 threads used
    // change this code:
    // first create a D_star matrix in shared memory - each thread independently
    // Then find the minimum of the D_star again using parallelism using parallel_red
    
    double min_distance = INT_MAX;
    for (int i = 0; i < num_taxa; i++) {
        if(dist_mat[i][0]!=-1) {
            for (int j = i + 1; j < num_taxa; j++) {
                if(dist_mat[j][0]!=-1){
                    double D_star = (num_taxa - 2) * dist_mat[i][j] - TD_arr[i] - TD_arr[j];
                    if (D_star < min_distance) {
                        min_distance = D_star;
                        index1 = i;
                        index2 = j;
                    }
                }
            }
        }
    }
    //return min_distance / (num_taxa - 2);
}

void updateDistanceMatrix(double dist_mat[MAX_TAXA][MAX_TAXA], int num_taxa, int min_index, int max_index) {
    // update the distance matrix parallely with new values
    for (int k = 0; k < num_taxa; k++) {
        if (k != min_index && k != max_index) {
            dist_mat[max_index][k] = ( dist_mat[min_index][k] + dist_mat[max_index][k] - dist_mat[min_index][max_index]) / 2;
            dist_mat[k][max_index] = dist_mat[max_index][k];
        }
    }
    dist_mat[min_index][0] = dist_mat[0][min_index] = -1;
}

__global__ void gpu_nj(num_taxa, dist_mat, TD_arr, temp_dist_mat){

    /*
        Device variables needed - dist_mat, TD_arr
    */ 

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int t_row = tid / num_taxa;
    int t_col = tid % num_taxa;
    __shared__ int index1, index2;
    __shared__ int min_index, max_index;
    __shared__ double delta_ij, limb_length_i, limb_length_j;
    int n;
    int i;
    double sum, min_d_star_row;
    int new_node_name;
    Node* temp_node;

    __shared__ double D_star_mat[100][2]; // declare in shared memory later

    // load dist_mat in shared memory
    // load TD_arr in shared memory
    
   
    // OPT - can go down the column per thread
    // parallel sum possible 
    for(int i=0 ; i<num_taxa-2; i++) {
        n = num_taxa - i;
        //totalDistance(dist_mat, num_taxa, TD_arr);
        // GPU implementation of totalDistance
        if(tid < num_taxa) {
            if(dist_mat[tid][0] != -1) {
                sum=0;
                for (int k = 0; k < num_taxa; k++) {
                    if(dist_mat[k][0] != -1){
                        sum += dist_mat[tid][k];
                    }
                }
                TD_arr[tid] = sum;
            } else{
                TD_arr[tid] = -1;
            }
        }


        //find_closest_pair(dist_mat,num_taxa, TD_arr, index1, index2);
        // GPU code for find_closest_pair
        min_d_star_row = INT_MAX;
        if(tid < num_taxa) {
            if(dist_mat[tid][0] != -1) {
                for (int j = tid + 1; j < num_taxa; j++) {
                    if(dist_mat[j][0] != -1){
                        min_d_star_row = (num_taxa - 2) * dist_mat[tid][j] - TD_arr[tid] - TD_arr[j];
                        if (min_d_star_row < D_star_mat[tid][0]) {
                            D_star_mat[tid][0] = min_d_star_row;
                            D_star_mat[tid][1] = j;
                            
                            }
                        }
                    }
                } else {
                    D_star_mat[tid][0] = INT_MAX;
                }
            }
        }

        // find the index pair which has absolute min among the d_star
        if(tid == 0) {
            min_d_star_row = INT_MAX;
            for (i = 0; i < num_taxa; i++) {
                if(D_star_mat[tid][0] < min_d_star_row){
                    min_d_star_row = D_star_mat[tid][0];
                    index1 = i;
                    index2 = D_star_mat[tid][1];
                }
            }
        }


        if(tid == 0) {
            min_index = index1 < index2 ? index1 : index2;
            max_index = index1 < index2 ? index2 : index1;
            delta_ij = (TD_arr[min_index] - TD_arr[max_index]) / (n-2);
            limb_length_i = (dist_mat[min_index][max_index] + delta_ij)/2.0;
            limb_length_j = (dist_mat[min_index][max_index] - delta_ij)/2.0;
        }


        //updateDistanceMatrix(dist_mat,num_taxa, min_index, max_index);

        // update the distance matrix parallely with new values
       
        // create a new distance matrix and swap the pointers 
        if((t_row != min_index) && (t_col !=  min_index) && (t_row != max_index) && (t_col !=  max_index)) {
                temp_dist_mat[t_row][t_col] = (dist_mat[min_index][t_col] + dist_mat[max_index][t_col] - dist_mat[min_index][max_index]) / 2;
        temp_dist_mat[min_index][0] = dist_mat[0][min_index] = -1;
        }
        if(tid == 0){
            temp_dist_mat[min_index][0] = -1;
            temp_dist_mat[0][min_index] = -1;
            dist_mat = temp_dist_mat;
   
         
            new_node_name = i;
            temp_node = Node_new_all(new_node_name, nodes[min_index], nodes[max_index], limb_length_i, limb_length_j );
            nodes[max_index] = temp_node;
            nodes[min_index] = nullptr;
        }
        __syncthreads();
    }

};



int main() {
    
    string file_name = "./examples/INGI2368.in";
    double dist_mat[MAX_TAXA][MAX_TAXA];
    char seq[MAX_TAXA];
    Node* nodes[MAX_TAXA];
    //int num_taxa = read_DM_file(dist_mat, seq, file_name, nodes);
    int num_taxa = readFromFile(dist_mat, seq, file_name, nodes);
    printDistanceMatrix(dist_mat, num_taxa, nodes);
    int index1, index2;
    int min_index, max_index;
    double delta_ij, limb_length_i, limb_length_j;
    int n;
    double TD_arr[MAX_TAXA];

    // Parallelize GPU
    for(int i=0 ; i<num_taxa-2; i++) {
        n = num_taxa - i;
        totalDistance(dist_mat, num_taxa, TD_arr);
        printTDMatrix(TD_arr, num_taxa);
        find_closest_pair(dist_mat,num_taxa, TD_arr, index1, index2);
        
        min_index = min(index1, index2);
        max_index = max(index1, index2);
        delta_ij = (TD_arr[min_index] - TD_arr[max_index]) / (n-2);
        limb_length_i = (dist_mat[min_index][max_index] + delta_ij)/2.0;
        limb_length_j = (dist_mat[min_index][max_index] - delta_ij)/2.0;
        updateDistanceMatrix(dist_mat,num_taxa, min_index, max_index);
        int new_node_name = i;
        cout<<to_string(new_node_name)<<endl;
        Node* temp = Node_new_all(new_node_name, nodes[min_index], nodes[max_index], limb_length_i, limb_length_j );
        nodes[max_index] = temp;
        nodes[min_index] = nullptr;
        printDistanceMatrix(dist_mat, num_taxa, nodes);
    }



    int final_index1 = -1;
    int final_index2 = -1;

    int i;
    for(i=0 ; i<num_taxa ; i++) {
        if(dist_mat[i][0]!=-1)
        {
            if(final_index1==-1)
                final_index1 = i;
            else
                final_index2 = i;
        }
    } 

    int root_node_name = i;
    cout<<to_string(root_node_name)<<endl;
    Node* root = Node_new_all(root_node_name, nodes[final_index1], nodes[final_index2], dist_mat[final_index1][final_index2]/2.0, dist_mat[final_index1][final_index2]/2.0 );

    // cout<<nodes[0]->node_name<<" "<<nodes[1]->node_name;
    
    ofstream outfile("g.gv"); // open the output file
    if (!outfile) {
        cerr << "Error opening file" << endl;
        exit(1);
    }
    outfile << "digraph {" << endl;
    traverseAndWrite(root, outfile);
    outfile << "}" << endl;
    outfile.close();


    return 0;
}
